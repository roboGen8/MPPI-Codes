#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays

__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

__global__
void fill1D(int length, double matrix[], double num) {
    for (int i = 0; i < length; i++) {
        matrix[i] = num;
    }
}

__global__
void fill2D(double* devPtr, size_t pitch, int width, int height) {
    for (int r = 0; r < height; ++r) {
        double * row = (double*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c) {
            double element = row[c];
            // printf("%f", row[c]);
        }
    }
}

__global__ void fill3D(hipPitchedPtr devPitchedPtr, int width, int height, int depth) {
    char* devPtr = (char*)devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height;
    for (int z = 0; z < depth; ++z) {
        char* slice = devPtr + z * slicePitch;
        for (int y = 0; y < height; ++y) {
            double* row = (double*)(slice + y * pitch);
            for (int x = 0; x < width; ++x) {
                double element = row[x];
            }
        }
    }
}

int main(void)
{
    //some variables to add values
    double *rb, *ra, *rp, *rc;
    hipMallocManaged(&rb, 400 * sizeof(double));
    hipMallocManaged(&ra, 400 * sizeof(double));
    hipMallocManaged(&rp, 400 * sizeof(double));
    hipMallocManaged(&rc, 400 * sizeof(double));


    fill1D<<<1, 256>>>(400, rb, 0.1);
    hipDeviceSynchronize();
    fill1D<<<1, 256>>>(400, ra, 0.1);
    hipDeviceSynchronize();
    fill1D<<<1, 256>>>(400, rp, 0.1);
    hipDeviceSynchronize();
    fill1D<<<1, 256>>>(400, rc, 0.1);
    hipDeviceSynchronize();

    // Host code
    //width is columns and height is rows
    int height = 400, width = 100, depth = 4;
    //2D stuff
    size_t pitch;

    double* delbmax;
    hipMallocPitch(&delbmax, &pitch, width * sizeof(double), height);
    // fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* delamax;
    hipMallocPitch(&delamax, &pitch, width * sizeof(double), height);

    double* delpmax;
    hipMallocPitch(&delpmax, &pitch, width * sizeof(double), height);

    double* delcmax;
    hipMallocPitch(&delcmax, &pitch, width * sizeof(double), height);

    double* ddelb;
    hipMallocPitch(&ddelb, &pitch, width * sizeof(double), height);

    double* ddela;
    hipMallocPitch(&ddela, &pitch, width * sizeof(double), height);

    double* ddelp;
    hipMallocPitch(&ddelp, &pitch, width * sizeof(double), height);

    double* ddelc;
    hipMallocPitch(&ddelc, &pitch, width * sizeof(double), height);

    double* del_bi;
    hipMallocPitch(&del_bi, &pitch, width * sizeof(double), height);

    double* del_ai;
    hipMallocPitch(&del_ai, &pitch, width * sizeof(double), height);

    double* del_pi;
    hipMallocPitch(&del_pi, &pitch, width * sizeof(double), height);

    double* del_ci;
    hipMallocPitch(&del_ci, &pitch, width * sizeof(double), height);

    //3D stuff
    hipExtent extent = make_hipExtent(width * sizeof(double), height, depth);
    hipPitchedPtr del_con;
    hipMalloc3D(&del_con, extent);
    fill3D<<<100, 40000>>>(del_con, width, height, depth);




    for (int j = 2; j <= 400; j++) {
        n = 100; //Counter of trajectories with final x-position outside of specified band in relation to ship's landing position
        m=1;    //Counter for number of cycles in while loop


        while (n > 90 && m < 10) {
            delbmax = rb[j - 1] * abs(del_con)

        }
    }

    hipFree(rb);
    hipFree(ra);
    hipFree(rp);
    hipFree(rc);

    hipFree(delbmax);
    hipFree(delamax);
    hipFree(delpmax);
    hipFree(delcmax);

    hipFree(ddelb);
    hipFree(ddela);
    hipFree(ddelp);
    hipFree(ddelc);

    hipFree(del_bi);
    hipFree(del_ai);
    hipFree(del_pi);
    hipFree(del_ci);

    hipFree(del_con)

    return 0;
}
