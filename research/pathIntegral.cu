
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays

__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

__global__
void fill1D(int length, double matrix[], double num) {
    for (int i = 0; i < length; i++) {
        matrix[i] = num;
    }
}

__global__
void fill2D(double* delbmax, size_t pitch, int width, int height) {
    for (int r = 0; r < height; ++r) {
        double * row = (double*)((char*)delbmax + r * pitch);
        for (int c = 0; c < width; ++c) {
            double element = row[c];
            // printf("%f", row[c]);
        }
    }
}

int main(void)
{
    //some variables to add values
    double *rb, *ra, *rp, *rc;
    hipMallocManaged(&rb, 400 * sizeof(double));
    hipMallocManaged(&ra, 400 * sizeof(double));
    hipMallocManaged(&rp, 400 * sizeof(double));
    hipMallocManaged(&rc, 400 * sizeof(double));


    fill1D<<<1, 256>>>(400, rb, 0.1);
    hipDeviceSynchronize();
    fill1D<<<1, 256>>>(400, ra, 0.1);
    hipDeviceSynchronize();
    fill1D<<<1, 256>>>(400, rp, 0.1);
    hipDeviceSynchronize();
    fill1D<<<1, 256>>>(400, rc, 0.1);
    hipDeviceSynchronize();

    // Host code
    //width is columns and height is rows
    int height = 400, width = 100;
    size_t pitch;

    double* delbmax;
    hipMallocPitch(&delbmax, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* delamax;
    hipMallocPitch(&delamax, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* delpmax;
    hipMallocPitch(&delpmax, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* delcmax;
    hipMallocPitch(&delcmax, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* ddelb;
    hipMallocPitch(&ddelb, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* ddela;
    hipMallocPitch(&ddela, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* ddelp;
    hipMallocPitch(&ddelp, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* ddelc;
    hipMallocPitch(&ddelc, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* del_bi;
    hipMallocPitch(&del_bi, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* del_ai;
    hipMallocPitch(&del_ai, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* del_pi;
    hipMallocPitch(&del_pi, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);

    double* del_ci;
    hipMallocPitch(&del_ci, &pitch, width * sizeof(double), height);
    fill2D<<<100, 400000>>>(delbmax, pitch, width, height);





    // for (int j = 2; j <= 400; j++) {
    //     n = 100; //Counter of trajectories with final x-position outside of specified band in relation to ship's landing position
    //     m=1;    //Counter for number of cycles in while loop
    //
    //
    //     while (n > 90 && m < 10) {
    //         delbmax = rb[j -1] * abs(del_con)
    //
    //     }
    // }

    hipFree(rb);
    hipFree(ra);
    hipFree(rp);
    hipFree(rc);

    hipFree(delbmax);

    return 0;
}
